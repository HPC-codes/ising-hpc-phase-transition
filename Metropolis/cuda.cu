#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <iomanip>
#include <fstream>

// Parámetros ajustados para mejor precisión
#define L 32
#define N (L*L)
#define J 1.0f
#define THERMALIZATION 100000  // Pasos de termalización
#define ITERATIONS 1000000     // Iteraciones de medición
#define NTHREADS 256
#define TEMP_START 1.0f
#define TEMP_END 3.0f
#define TEMP_STEP 0.1f
#define MEASUREMENT_INTERVAL 100

__global__ void setup_rand_kernel(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

__global__ void initialize_lattice_kernel(int* lattice, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        lattice[idx] = (hiprand_uniform(&states[idx]) < 0.5f) ? -1 : 1;
    }
}

__device__ int get_index(int row, int col) {
    return (row * L + col);
}

__device__ float calculate_energy_change(int* lattice, int idx, float T) {
    int row = idx / L;
    int col = idx % L;
    
    int up = get_index((row - 1 + L) % L, col);
    int down = get_index((row + 1) % L, col);
    int left = get_index(row, (col - 1 + L) % L);
    int right = get_index(row, (col + 1) % L);
    
    int spin = lattice[idx];
    int sum_neighbors = lattice[up] + lattice[down] + lattice[left] + lattice[right];
    return 2.0f * J * spin * sum_neighbors;
}

__global__ void metropolis_sweep(int* lattice, float* total_energy, hiprandState* states, float T, bool even_sweep) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    int row = idx / L;
    int col = idx % L;
    
    // Actualización checkerboard
    if ((row + col) % 2 == even_sweep) {
        float delta_E = calculate_energy_change(lattice, idx, T);
        float rnd = hiprand_uniform(&states[idx]);
        
        if (delta_E <= 0.0f || rnd < __expf(-delta_E / T)) {
            lattice[idx] = -lattice[idx];
            atomicAdd(total_energy, 2.0f * delta_E);
        }
    }
}

__global__ void calculate_observables(int* lattice, float* magnetization, float* energy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int spin = lattice[idx];
        atomicAdd(magnetization, spin);
        
        // Calcular energía solo para una dirección para evitar doble conteo
        if (idx % L != L - 1) {  // Vecino derecho
            atomicAdd(energy, -J * spin * lattice[idx + 1]);
        }
        if (idx / L != L - 1) {  // Vecino inferior
            atomicAdd(energy, -J * spin * lattice[idx + L]);
        }
    }
}

int main() {
    // Configuración CUDA
    dim3 blocks((N + NTHREADS - 1) / NTHREADS);
    dim3 threads(NTHREADS);
    
    // Asignación de memoria
    int* d_lattice;
    float* d_magnetization;
    float* d_energy;
    hiprandState* d_states;
    
    hipMalloc((void**)&d_lattice, N * sizeof(int));
    hipMalloc((void**)&d_magnetization, sizeof(float));
    hipMalloc((void**)&d_energy, sizeof(float));
    hipMalloc((void**)&d_states, N * sizeof(hiprandState));
    
    // Inicialización
    setup_rand_kernel<<<blocks, threads>>>(d_states, time(nullptr));
    initialize_lattice_kernel<<<blocks, threads>>>(d_lattice, d_states);
    
    // Archivo de resultados
    std::ofstream results("ising_results.csv");
    results << "Temperature,Magnetization,Energy,Susceptibility,SpecificHeat,BinderCumulant\n";
    results << std::scientific << std::setprecision(6);
    
    // Bucle de temperatura
    for (float T = TEMP_START; T <= TEMP_END; T += TEMP_STEP) {
        clock_t start = clock();
        
        // Termalización
        for (int i = 0; i < THERMALIZATION; i++) {
            float zero = 0.0f;
            hipMemcpy(d_energy, &zero, sizeof(float), hipMemcpyHostToDevice);
            
            metropolis_sweep<<<blocks, threads>>>(d_lattice, d_energy, d_states, T, true);
            metropolis_sweep<<<blocks, threads>>>(d_lattice, d_energy, d_states, T, false);
        }
        
        // Mediciones
        float sum_M = 0.0f, sum_M2 = 0.0f, sum_M4 = 0.0f;
        float sum_E = 0.0f, sum_E2 = 0.0f;
        int measurements = 0;
        
        for (int i = 0; i < ITERATIONS; i++) {
            float zero = 0.0f;
            hipMemcpy(d_energy, &zero, sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_magnetization, &zero, sizeof(float), hipMemcpyHostToDevice);
            
            metropolis_sweep<<<blocks, threads>>>(d_lattice, d_energy, d_states, T, true);
            metropolis_sweep<<<blocks, threads>>>(d_lattice, d_energy, d_states, T, false);
            
            if (i % MEASUREMENT_INTERVAL == 0) {
                calculate_observables<<<blocks, threads>>>(d_lattice, d_magnetization, d_energy);
                hipDeviceSynchronize();
                
                float M, E;
                hipMemcpy(&M, d_magnetization, sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(&E, d_energy, sizeof(float), hipMemcpyDeviceToHost);
                
                M /= N;
                E /= N;
                
                sum_M += fabs(M);
                sum_M2 += M * M;
                sum_M4 += M * M * M * M;
                sum_E += E;
                sum_E2 += E * E;
                measurements++;
            }
        }
        
        // Cálculo de promedios
        float avg_M = sum_M / measurements;
        float avg_M2 = sum_M2 / measurements;
        float avg_M4 = sum_M4 / measurements;
        float avg_E = sum_E / measurements;
        float avg_E2 = sum_E2 / measurements;
        
        float var_M = (avg_M2 - avg_M * avg_M) * N;
        float var_E = (avg_E2 - avg_E * avg_E) * N;
        
        float susceptibility = var_M / T;
        float specific_heat = var_E / (T * T);
        float binder_cumulant = 1.0f - avg_M4 / (3.0f * avg_M2 * avg_M2);
        
        // Guardar resultados
        results << T << "," << avg_M << "," << avg_E << "," 
               << susceptibility << "," << specific_heat << "," 
               << binder_cumulant << "\n";
        
        // Mostrar progreso
        clock_t end = clock();
        double elapsed = double(end - start) / CLOCKS_PER_SEC;
        
        std::cout << "T = " << std::setw(4) << T 
                 << "  |M| = " << std::setw(8) << avg_M 
                 << "  E = " << std::setw(8) << avg_E
                 << "  χ = " << std::setw(8) << susceptibility
                 << "  C = " << std::setw(8) << specific_heat
                 << "  U = " << std::setw(8) << binder_cumulant
                 << "  [" << elapsed << "s]\n";
    }
    
    // Liberar memoria
    hipFree(d_lattice);
    hipFree(d_magnetization);
    hipFree(d_energy);
    hipFree(d_states);
    
    results.close();
    std::cout << "\nResultados guardados en ising_results.csv\n";
    
    return 0;
}